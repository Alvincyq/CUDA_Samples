#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <malloc.h>
#include <random>
#include <time.h>

const int threadPerBlock = 16;

texture<int> texA;
texture<int> texB;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

hipError_t mulWithCuda(const int *a, const int *b, int *result, const int M, const int N, const int S);

hipError_t mulWithCudaTex(const int *a, const int *b, int *result, const int M, const int N, const int S);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

/* MatMultiply��CPU�¾���˷�
*  a:��һ������ָ�룬��ʾa[M][N];
*  b:�ڶ�������ָ�룬��ʾb[N][S];
*  result:������󣬱�ʾΪresult[M][S];
*/
void CPUMatMultiply(const int * a,const int * b, int *result,const int M,const int N,const int S)
{
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			int index = i * S + j;
			result[index] = 0;

			//����ÿһ��Ԫ�صĽ��
			for (int k = 0; k < N; k++)
			{
				result[index] += a[i * N + k] * b[k * S + j];
			}
		}
	}
}

/* gpuMatMultKernel��GPU�¾���˷��˺���
*  a:��һ������ָ�룬��ʾa[M][N]
*  b:�ڶ�������ָ�룬��ʾb[N][S]
*  result:������󣬱�ʾresult[M][S]
*/
__global__ void gpuMatMultKernel(const int *a, const int *b, int *result, const int M, const int N, const int S)
{
	//int threadId = threadIdx.x + blockIdx.x * blockDim.x;

	int threadId = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < M * S)
	{
		int row = threadId / S;
		int column = threadId % S;

		result[threadId] = 0;
		for (int i = 0; i < N; i++)
		{
			result[threadId] += a[row * N + i] * b[i * S + column];
		}
	}
}

/* gpuMatMultWithSharedKernel��GPU��ʹ��shared�ڴ�ľ���˷�
*  a:��һ������ָ�룬��ʾa[height_A][width_A]
*  b:�ڶ�������ָ�룬��ʾb[width_A][width_B]
*  result:������󣬱�ʾresult[height_A][width_B]
*/
template<int BLOCK_SIZE>
__global__ void gpuMatMultWithSharedKernel(const int *a, const int *b, int *result, const int height_A, const int width_A, const int width_B)
{
	int block_x = blockIdx.x;
	int block_y = blockIdx.y;
	int thread_x = threadIdx.x;
	int thread_y = threadIdx.y;

	if ((thread_y + block_y * blockDim.y) * width_B + block_x * blockDim.x + thread_x >= height_A * width_B)
	{
		return;
	}

	const int begin_a = block_y * blockDim.y * width_A;
	const int end_a = begin_a + width_A - 1;
	const int step_a = blockDim.x;

	const int begin_b = block_x * blockDim.x;
	const int step_b = blockDim.y * width_B;

	int result_temp = 0;

	for (int index_a = begin_a, int index_b = begin_b;
		index_a < end_a; index_a += step_a, index_b += step_b)
	{
		__shared__ int SubMat_A[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ int SubMat_B[BLOCK_SIZE][BLOCK_SIZE];

		SubMat_A[thread_y][thread_x] = a[index_a + thread_y * width_A + thread_x];
		SubMat_B[thread_y][thread_x] = b[index_b + thread_y * width_B + thread_x];

		__syncthreads();

		for (int i = 0; i < BLOCK_SIZE; i++)
		{
			result_temp += SubMat_A[thread_y][i] * SubMat_B[i][thread_x];
		}

		__syncthreads();
	}

	int begin_result = block_y * blockDim.y * width_B + begin_b;
	result[begin_result + thread_y * width_B + thread_x] = result_temp;
}

/* gpuMatMultWithTextureKernel��GPU��ʹ��texture�ڴ�ľ���˷�
*  result��������󣬱�ʾΪresult[M][S];
*  M����ʾΪ����A�����result������
*  N����ʾ����A������������B������
*  S����ʾ����B�;���result������
*/
__global__ void gpuMatMultWithTextureKernel(int * result, const int M, const int N, const int S)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	if (offset < M * S)
	{
		int a = 0, b = 0;
		int temp_result = 0;
		for (int i = 0; i < N; i++)
		{
			a = tex1Dfetch(texA, y * N + i);
			b = tex1Dfetch(texB, i * S + x);
			temp_result += a * b;
		}
		result[offset] = temp_result;
	}
}


// main���������ֱ�����CPU��GPU����˷��������Ƚ϶��ߵ�����ʱ��
int main()
{

	//ȷ������Ĵ�С
	int M = 0, N = 0, S = 0;
	printf("please input the value of M (Mat a's row):");
	scanf("%d", &M);
	printf("please input the value of N (Mat a's column and Mat b's row):");
	scanf("%d", &N);
	printf("please input the value of S (Mat b's column):");
	scanf("%d", &S);

	//�������ռ�
	int * a = (int *)malloc(M * N * sizeof(int));
	if (NULL == a)
	{
		printf("the malloc of Mat a is failed!\n");
		return 0;
	}
	int * b = (int *)malloc(N * S * sizeof(int));
	if (NULL == b)
	{
		printf("the malloc of Mat b is failed!\n");
		return 0;
	}
	//cpu��gpu�Ľ������ֱ���
	int * cpuResult = (int *)malloc(M * S * sizeof(int));
	if (NULL == cpuResult)
	{
		printf("the malloc of Mat cpuResult is failed!\n");
		return 0;
	}
	int * gpuResult = (int *)malloc(M * S * sizeof(int));
	if (NULL == cpuResult)
	{
		printf("the malloc of Mat gpuResult is failed!\n");
		return 0;
	}

	//���ɾ�������
	printf("\nstart random the Mat a...\n");
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < N; j++)
		{
			a[i * N + j] = rand() % 5;
		}
	}

	printf("\nstart random the Mat b...\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < S; j++)
		{
			b[i * S + j] = rand() % 5;
		}
	}

	//ͳ��CPU���г˷���ʱ��
	clock_t start, finish;
	double totalTime = 0.0;
	start = clock();

	//����CPU����˷�����
	CPUMatMultiply(a, b, cpuResult, M, N, S);

	finish = clock();
	totalTime = (double)(finish - start) / CLOCKS_PER_SEC;
	printf("\nThe total time is %lf seconds!\n", totalTime);

	//����GPU����˷�����
	hipError_t cudaStatus = mulWithCuda(a, b, gpuResult, M, N, S);
	//hipError_t cudaStatus = mulWithCudaTex(a, b, gpuResult, M, N, S);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "mulWithCuda failed!");
		return 0;
	}
	//��ӡ�������result
	/*printf("\nthe result of CPU :\n");
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			printf("%d\t", cpuResult[i * M + j]);
		}
		printf("\n");
	}

	printf("\nthe result of GPU :\n");
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			printf("%d\t", gpuResult[i * M + j]);
		}
		printf("\n");
	}*/

	//ȷ��CPU��GPU����˷�����Ƿ���ͬ���Ӷ�˵������Ƿ���ȷ
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			if (cpuResult[i * M + j] != gpuResult[i * M + j])
			{
				printf("the Result isn't equal!\n");
				return 0;
			}
		}
	}

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

// ����CUDA����GPU����˷��˺���
hipError_t mulWithCuda(const int *a, const int *b, int *result, const int M, const int N, const int S)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_result = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_a, M * N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_b, N * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_b failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_result, M * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_result failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a, a, M * N * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudamemcpy dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, N * S * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy dev_b failed!\n");
		goto Error;
	}

	hipEvent_t gpuStart, gpuFinish;
	float elapsedTime;
	hipEventCreate(&gpuStart);
	hipEventCreate(&gpuFinish);
	hipEventRecord(gpuStart, 0);

	/*const int THREADNUM = 256;
	const int BLOCKNUM = (M * S + 255) / 256;*/

	const int BLOCK_SIZE = 16;
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((S + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
	gpuMatMultKernel << <grid, block >> >(dev_a, dev_b, dev_result, M, N, S);
	//gpuMatMultWithSharedKernel<16> << <grid, block >> >(dev_a, dev_b, dev_result, M, N, S);

	hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	printf("\nThe runing time of GPU on Mat Multiply is %f seconds.\n", elapsedTime / 1000.0);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(result, dev_result, M * S * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy result failed!\n");
		goto Error;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_result);

	return cudaStatus;
}

//����CUDA����GPU����˷��˺���
//������A�����B�󶨵������ڴ���
hipError_t mulWithCudaTex(const int *a, const int *b, int *result, const int M, const int N, const int S)
{
	int * dev_a = 0;
	int * dev_b = 0;
	int * dev_result = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA_capable GPU installed?\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_a, M * N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_b, N * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_b failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_result, M * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_result failed!\n");
		goto Error;
	}

	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	cudaStatus = hipBindTexture(NULL, texA, dev_a, desc, M * N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipBindTexture texA failed!\n");
		goto Error;
	}

	cudaStatus = hipBindTexture(NULL, texB, dev_b, desc, N * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipBindTexture texB failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a, a, M * N * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudamemcpy dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, N * S * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy dev_b failed!\n");
		goto Error;
	}

	hipEvent_t gpuStart, gpuFinish;
	float elapsedTime;
	hipEventCreate(&gpuStart);
	hipEventCreate(&gpuFinish);
	hipEventRecord(gpuStart, 0);

	const int BLOCK_SIZE = 16;
	if ((M % BLOCK_SIZE != 0) && (S % BLOCK_SIZE != 0))
	{
		fprintf(stderr, "M or S can't be dividen by 16!\n");
		goto Error;
	}

	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(S / BLOCK_SIZE, M / BLOCK_SIZE);
	gpuMatMultWithTextureKernel << <grid, block >> >(dev_result, M, N, S);

	hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	printf("\nThe runing time of GPU on Mat Multiply is %f seconds.\n", elapsedTime / 1000.0);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(result, dev_result, M * S * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy result failed!\n");
		goto Error;
	}

Error:
	hipUnbindTexture(texA);
	hipUnbindTexture(texB);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_result);

	return cudaStatus;

}